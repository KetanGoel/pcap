#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <ctime>
#include <float.h>
#include "mypgm.h"

//kernel: function executed by GPU threads

__global__ void sobel_gpu(float* image1, float* image2, int w, int h) {

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    float dx, dy;

    if (x > 0 && y > 0 && x < w - 1 && y < h - 1){

        dx = (-1 * image1[(y - 1) * w + (x - 1)]) + (-2 * image1[y * w + (x - 1)]) + (-1 * image1[(y + 1) * w + (x - 1)]) +
            (image1[(y - 1) * w + (x + 1)]) + (2 * image1[y * w + (x + 1)]) + (image1[(y + 1) * w + (x + 1)]);

        dy = (image1[(y - 1) * w + (x - 1)]) + (2 * image1[(y - 1) * w + x]) + (image1[(y - 1) * w + ( x + 1)]) +
            (-1 * image1[(y + 1) * w + (x - 1)]) + (-2 * image1[(y + 1) * w + x]) + (-1 * image1[(y + 1 ) * w + (x + 1)]);

        image2[y * w + x] = sqrt((dx * dx) + (dy * dy));
    }
}
 
//host: a function that executes on the processor

__host__ void sobel_cpu(float image1[1024][1024], float image3[1024][1024], int w, int h) {

    float dx, dy;

    for (int i = 0; i < w-2; i++) {
        for (int j = 0; j < h-2; j++) {

           
                dy = (-1 * image1[i][j]) + (-2 * image1[i][j+1]) + (-1 * image1[i][j+2]) +
                    (image1[i+2][j]) + (2 * image1[i+2][j+1]) + (image1[i+2][j+2]);


                dx = (-1 * image1[i][j]) + (1 * image1[i][j + 2])
                    + (-2* image1[i+1][j]) + (2 * image1[i+1][j+2])
                    + (-1 * image1[i+2][j]) + (1 * image1[i+2][j+2]);

                image3[i][j] = sqrt((dx * dx) + (dy * dy));
        }
    }
}

int main() {

    //Load Image (image image1 is a global variable for simplicity)
    load_image_data();

    int i, j;

    float* deviceInputImageData;
    float* deviceOutputImageData;

    //Allocating memory on the graphics card
    hipMalloc((void**)&deviceInputImageData, x_size1 * y_size1 * sizeof(float));
    hipMalloc((void**)&deviceOutputImageData, x_size1 * y_size1 * sizeof(float));

    //copying values ​​from main memory to graphics card memory
    hipMemcpy(deviceInputImageData, image1, x_size1 * y_size1 * sizeof(float), hipMemcpyHostToDevice);

    //Initialize the clock so we can track GPU execution time
    std::clock_t start_gpu;
    double duration_gpu;
    start_gpu = std::clock();

    printf(" ----------- GPU -----------\n");

    //Filling matrix2 (new images)//
    printf("~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ ~~~~~~~~~~~~~~~~~~~~~~~~~~~\n");
    printf("INITIALIZE image2 (IMAGE_GPU)\n");
    x_size2 = x_size1;
    y_size2 = y_size1;
    for (i = 0; i < y_size2; i++) {
        for (j = 0; j < x_size2; j++) {
            image2[i][j] = 0;
        }
    }

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(64, 64);

    printf("\nBEGINNING: Sobel filter on GPU\n");
    //Function <<<dimGrid, dimBlock>>>
    sobel_gpu<<<numBlocks, threadsPerBlock>>>(deviceInputImageData, deviceOutputImageData, x_size1, y_size1);

    //End GPU clock
    duration_gpu = (std::clock() - start_gpu) / (double)CLOCKS_PER_SEC;
    std::cout << "\nCOMPLETE: Sobel filter on GPU: " << duration_gpu << "s - SUCCESS!" << '\n';
    printf("~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ ~~~~~~~~~~~~~~~~~~~~~~~~~~~\n\n\n");

    //copy the value from the graphics card to the main memory
    hipMemcpy(image2, deviceOutputImageData, x_size2 * y_size2 * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(deviceInputImageData);
    hipFree(deviceOutputImageData);

    save_image_data_img2();

    //initialize the clock so we can monitor CPU execution time
    std::clock_t start_cpu;
    double duration_cpu;
    start_cpu = std::clock();

    printf(" ----------- CPU -----------\n");

    printf("~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ ~~~~~~~~~~~~~~~~~~~~~~~~~~~\n");
    printf("INITIALIZE image3 (IMAGE_CPU)\n");
    x_size3 = x_size1;
    y_size3 = y_size1;
    for (i = 0; i < y_size3; i++) {
        for (j = 0; j < x_size3; j++) {
            image3[i][j] = 0;
        }
    }

    printf("\nBEGINNING: Sobel filter on CPU\n");

    sobel_cpu(image1, image3, x_size3, y_size3);

    ////End of clock CPU
    duration_cpu = (std::clock() - start_cpu) / (double)CLOCKS_PER_SEC;
    std::cout << "\nCOMPLETE: Sobel filter on CPU: " << duration_cpu << "s - SUCCESS!" << '\n';
    printf("~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ ~~ ~~~~~~~~~~~~~~~~~~~~~~~~~~~\n\n\n");

    //Difference in execution time:
    double duration;
    double times;

    duration = duration_cpu - duration_gpu;
    times = duration_cpu / duration_gpu * 100;

    printf("~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ ~~~~~~~~~~~~~~~~~~~~~~~~~~~");
    std::cout << "\nEXECUTION TIME DIFFERENCE CPU - GPU: " << duration << '\n';
    std::cout << "\nGPU is FASTER than CPU by: " << times << " %" << '\n';
    printf("~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ ~~~~~~~~~~~~~~~~~~~~~~~~~~~\n\n\n");
    
    save_image_data_img3();

    return 0;
}
